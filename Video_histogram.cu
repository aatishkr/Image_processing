#include "hip/hip_runtime.h"
__global__ void convolve(unsigned char *source, int width, int height, int paddingX, int paddingY, ssize_t kOffset, int kWidth, int kHeight, unsigned char *destination)
{
    // Calculate our pixel's location
    int x = (blockIdx.x * blockDim.x) + threadIdx.x;
    int y = (blockIdx.y * blockDim.y) + threadIdx.y;

    float sum = 0.0;
    int   pWidth = kWidth/2;
    int   pHeight = kHeight/2;

    // Only execute for valid pixels
    if(x >= pWidth+paddingX &&
       y >= pHeight+paddingY &&
       x < (blockDim.x * gridDim.x)-pWidth-paddingX &&
       y < (blockDim.y * gridDim.y)-pHeight-paddingY)
    {
        for(int j = -pHeight; j <= pHeight; j++)
        {
            for(int i = -pWidth; i <= pWidth; i++)
            {
                // Sample the weight for this location
                int ki = (i+pWidth);
                int kj = (j+pHeight);
                float w  = convolutionKernelStore[(kj * kWidth) + ki + kOffset];

        
                sum += w * float(source[((y+j) * width) + (x+i)]);
            }
        }
    }
    